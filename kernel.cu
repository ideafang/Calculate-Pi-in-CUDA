#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdio.h>
#include <ctime>
#include <vector>


using namespace std;

clock_t c_start, c_end;
int n = 1024 * 1024 * 32;

__global__ void count_pi_1(float *dev_randX, float *dev_randY, int *dev_threads_num, int n) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	int cont = 0;
	for (int i = tid * 128; i < 128 * (tid + 1); i++) {
		if (dev_randX[i] * dev_randX[i] + dev_randY[i] * dev_randY[i] < 1.0f) {
			cont++;
		}
	}
	dev_threads_num[tid] = cont;
}

__global__ void count_pi_2(float *dev_randX, float *dev_randY, int *dev_blocks_num, int n) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	__shared__ int count_pi_2[512];
	int cont = 0;
	for (int i = tid * 128; i < 128 * (tid + 1); i++) {
		if (dev_randX[i] * dev_randX[i] + dev_randY[i] * dev_randY[i] < 1.0f) {
			cont++;
		}
	}
	count_pi_2[threadIdx.x] = cont;

	__syncthreads();

	if (threadIdx.x == 0) {
		int total = 0;
		for (int j = 0; j < 512; j++) {
			total += count_pi_2[j];
		}
		dev_blocks_num[blockIdx.x] = total;
	}
}

__global__ void count_pi_3(float *dev_randX, float *dev_randY, int *dev_blocks_num, int n) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = gridDim.x * blockDim.x;
	__shared__ int count_pi_3[512];
	int cont = 0;
	for (int i = tid; i < n; i += stride) {
		if (dev_randX[i] * dev_randX[i] + dev_randY[i] * dev_randY[i] < 1.0f) {
			cont++;
		}
	}
	count_pi_3[threadIdx.x] = cont;

	__syncthreads();

	if (threadIdx.x == 0) {
		int total = 0;
		for (int j = 0; j < 512; j++) {
			total += count_pi_3[j];
		}
		dev_blocks_num[blockIdx.x] = total;
	}
}


__global__ void count_pi_4(float *dev_randX, float *dev_randY, int *dev_threads_num, int n) {
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = gridDim.x * blockDim.x;
	int cont = 0;
	for (int i = tid; i < n; i += stride) {
		if (dev_randX[i] * dev_randX[i] + dev_randY[i] * dev_randY[i] < 1.0f) {
			cont++;
		}
	}
	dev_threads_num[tid] = cont;
}


int main() {

	vector<float> randX(n);
	vector<float> randY(n);

	srand((unsigned)time(NULL));
	for (int i = 0; i < n; i++) {
		randX[i] = float(rand()) / RAND_MAX;
		randY[i] = float(rand()) / RAND_MAX;
	}
	//start cont cpu time
	c_start = clock();
	int c_count = 0;
	//CPU calculate pi
	for (int i = 0; i < n; i++) {
		if (randX[i] * randX[i] + randY[i] * randY[i] < 1.0f) {
			c_count++;
		}
	}
	//end cont cpu time
	c_end = clock();
	float t_cpu = (float)(c_end - c_start) / CLOCKS_PER_SEC;
	float c_num = float(c_count) * 4.0 / n;
	cout << "CPU Time" << endl;
	cout << c_num << endl;
	cout << "time= " << t_cpu * 1000 << " ms" << endl;


	//start cont gpu time
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//send data to GPU
	size_t size = n * sizeof(float);
	float *dev_randX;
	float *dev_randY;
	hipMalloc((void**)&dev_randX, size);
	hipMalloc((void**)&dev_randY, size);

	hipMemcpy(dev_randX, &randX.front(), size, hipMemcpyHostToDevice);
	hipMemcpy(dev_randY, &randY.front(), size, hipMemcpyHostToDevice);

	int threadsPerBlock = 512;
	int block_num = n / (128 * threadsPerBlock);
	int *dev_threads_num;
	hipMalloc((void**)&dev_threads_num, n / 128 * sizeof(int));

	//调用GPU计算
	count_pi_1 <<<block_num, threadsPerBlock >>> (dev_randX, dev_randY, dev_threads_num, n);

	//计算时间及pi值
	int* threads_num = new int[n / 128];
	hipMemcpy(threads_num, dev_threads_num, n / 128 * sizeof(int), hipMemcpyDeviceToHost);

	int g_count = 0;
	for (int i = 0; i < n / 128; i++) {
		g_count += threads_num[i];
	};

	//end cont gpu time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float t_gpu1;
	hipEventElapsedTime(&t_gpu1, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	float g_num = float(g_count) * 4.0 / n;
	cout << "GPU_1 Time" << endl;
	cout << g_num << endl;
	cout << "time = " << t_gpu1 << " ms" << endl;

	//count_pi_1结束，count_pi_2开始

	//start cont gpu time
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	int *dev_blocks_num;
	hipMalloc((void**)&dev_blocks_num, 512 * sizeof(int));

	hipMemcpy(dev_randX, &randX.front(), size, hipMemcpyHostToDevice);
	hipMemcpy(dev_randY, &randY.front(), size, hipMemcpyHostToDevice);

	//调用GPU计算
	count_pi_2 << <block_num, threadsPerBlock >> > (dev_randX, dev_randY, dev_blocks_num, n);

	//计算时间及pi值
	int *blocks_num = new int[block_num];
	hipMemcpy(blocks_num, dev_blocks_num, block_num * sizeof(int), hipMemcpyDeviceToHost);

	g_count = 0;
	for (int i = 0; i < block_num; i++) {
		g_count += blocks_num[i];
	};

	//end cont gpu time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float t_gpu2;
	hipEventElapsedTime(&t_gpu2, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	g_num = float(g_count) * 4.0 / n;
	cout << "GPU_2 Time(共享内存)" << endl;
	cout << g_num << endl;
	cout << "time = " << t_gpu2 << " ms" << endl;

	//count_pi_2结束，count_pi_3开始

	//start cont gpu time
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipMemset(dev_blocks_num, 0, sizeof(int));

	hipMemcpy(dev_randX, &randX.front(), size, hipMemcpyHostToDevice);
	hipMemcpy(dev_randY, &randY.front(), size, hipMemcpyHostToDevice);

	//调用GPU计算
	count_pi_3 << <block_num, threadsPerBlock >> > (dev_randX, dev_randY, dev_blocks_num, n);

	//计算时间及pi值
	blocks_num = new int[block_num];
	hipMemcpy(blocks_num, dev_blocks_num, block_num * sizeof(int), hipMemcpyDeviceToHost);

	g_count = 0;
	for (int i = 0; i < block_num; i++) {
		g_count += blocks_num[i];
	};

	//end cont gpu time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float t_gpu3;
	hipEventElapsedTime(&t_gpu3, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	g_num = float(g_count) * 4.0 / n;
	cout << "GPU_3 Time(合并访问)" << endl;
	cout << g_num << endl;
	cout << "time = " << t_gpu3 << " ms" << endl;

	//count_pi_3结束，count_pi_4开始

	//start cont gpu time
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	hipMemset(dev_threads_num, 0, sizeof(int));

	hipMemcpy(dev_randX, &randX.front(), size, hipMemcpyHostToDevice);
	hipMemcpy(dev_randY, &randY.front(), size, hipMemcpyHostToDevice);
	//调用GPU计算
	count_pi_4 << <block_num, threadsPerBlock >> > (dev_randX, dev_randY, dev_threads_num, n);

	//计算时间及pi值
	threads_num = new int[n / 128];
	hipMemcpy(threads_num, dev_threads_num, n / 128 * sizeof(int), hipMemcpyDeviceToHost);

	g_count = 0;
	for (int i = 0; i < n / 128; i++) {
		g_count += threads_num[i];
	};

	//end cont gpu time
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float t_gpu4;
	hipEventElapsedTime(&t_gpu4, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	g_num = float(g_count) * 4.0 / n;
	cout << "GPU_4 Time" << endl;
	cout << g_num << endl;
	cout << "time = " << t_gpu4 << " ms" << endl;

	hipFree(dev_randX);
	hipFree(dev_randY);
	hipFree(dev_threads_num);
	hipFree(dev_blocks_num);
}